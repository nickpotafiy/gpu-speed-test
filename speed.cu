#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { \
    hipError_t err = (x); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        throw std::runtime_error("CUDA error"); \
    } \
} while (0)

#define CURAND_CALL(x) do { \
    hiprandStatus_t err = (x); \
    if (err != HIPRAND_STATUS_SUCCESS) { \
        std::cerr << "CURAND Error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        throw std::runtime_error("CURAND error"); \
    } \
} while (0)

#define CUDA_DEVICE(x) do { \
    hipError_t err = hipSetDevice(x); \
    if (err != hipSuccess) { \
        if (err == hipErrorInvalidDevice) { \
            std::cerr << "CUDA Error: Invalid Device " << x << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        } else { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        } \
        throw std::runtime_error("Device does not exist"); \
    } \
} while(0)

void enablePeerAccess(int gpuFrom, int gpuTo) {
    int canAccessPeer;
    CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, gpuFrom, gpuTo));
    if (canAccessPeer) {
        CUDA_CALL(hipSetDevice(gpuFrom));
        hipError_t err = hipDeviceEnablePeerAccess(gpuTo, 0);
        if (err == hipErrorPeerAccessAlreadyEnabled) {
            err = hipSuccess;
        }
        CUDA_CALL(err);

        CUDA_CALL(hipSetDevice(gpuTo));
        err = hipDeviceEnablePeerAccess(gpuFrom, 0);
        if (err == hipErrorPeerAccessAlreadyEnabled) {
            err = hipSuccess;
        }
        CUDA_CALL(err);
    }
}

float* allocateTestData(int gpu, size_t size, bool fill = false) {
    float* data = nullptr;
    CUDA_DEVICE(gpu);
    CUDA_CALL(hipMalloc((void**)&data, size * sizeof(float)));
    
    if(fill) {
        hiprandGenerator_t gen;
        CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
        CURAND_CALL(hiprandGenerateUniform(gen, data, size));
        CURAND_CALL(hiprandDestroyGenerator(gen));
    }
    
    return data;
}

void moveTestData(float* gpu1Data, float* gpu2Data, int gpuFrom, int gpuTo, size_t size) {
    enablePeerAccess(gpuFrom, gpuTo);
    
    CUDA_CALL(hipSetDevice(gpuFrom));
    
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    
    CUDA_CALL(hipEventRecord(start, 0));    
    CUDA_CALL(hipMemcpyPeer(gpu2Data, gpuTo, gpu1Data, gpuFrom, size * sizeof(float)));    
    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    std::cout << "Copied " << ((size / 1024 / 1024) * 4) << " MiB from GPU " << gpuFrom << " to GPU " << gpuTo << " in " << milliseconds << " ms" << std::endl;
    
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

}

void test(int gpu0, int gpu1) {

    const size_t dataSize = 1024 * 1024 * 1024 / 4;

    float* gpu0Data = allocateTestData(gpu0, dataSize, true);
    float* gpu1Data = allocateTestData(gpu1, dataSize, false);

    moveTestData(gpu0Data, gpu1Data, gpu0, gpu1, dataSize);

    CUDA_CALL(hipFree(gpu0Data));
    CUDA_CALL(hipFree(gpu1Data));
}

int main() {

    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return -1;
    }

    std::cout << "Total devices found:" << num_devices << std::endl;
    for(int i = 0; i < num_devices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << std::endl << "[ Device " << i << ": " << prop.name << "]" << std::endl << std::endl;
        for(int j = 0; j < num_devices; j++) {
            if(i != j) {
                test(i, j);
            }
        }
    }
    return 0;
}