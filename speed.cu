#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#include <hiprand/hiprand_kernel.h>
#include <string>

#define CUDA_CALL(x) do { \
    hipError_t err = (x); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        throw std::runtime_error("CUDA error"); \
    } \
} while (0)

#define CURAND_CALL(x) do { \
    hiprandStatus_t err = (x); \
    if (err != HIPRAND_STATUS_SUCCESS) { \
        std::cerr << "CURAND Error at " << __FILE__ << ":" << __LINE__ << std::endl; \
        throw std::runtime_error("CURAND error"); \
    } \
} while (0)

#define CUDA_DEVICE(x) do { \
    hipError_t err = hipSetDevice(x); \
    if (err != hipSuccess) { \
        if (err == hipErrorInvalidDevice) { \
            std::cerr << "CUDA Error: Invalid Device " << x << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        } else { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        } \
        throw std::runtime_error("Device does not exist"); \
    } \
} while(0)

std::string formatDouble(double value, int precision) {
    char buffer[100];
    char format[10];
    snprintf(format, sizeof(format), "%%.%df", precision);
    snprintf(buffer, sizeof(buffer), format, value);
    return std::string(buffer);
}

void enablePeerAccess(int gpuFrom, int gpuTo) {
    int canAccessPeer;
    CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, gpuFrom, gpuTo));
    if (canAccessPeer) {
        CUDA_CALL(hipSetDevice(gpuFrom));
        hipError_t err = hipDeviceEnablePeerAccess(gpuTo, 0);
        if (err == hipErrorPeerAccessAlreadyEnabled) {
            err = hipSuccess;
        }
        CUDA_CALL(err);

        CUDA_CALL(hipSetDevice(gpuTo));
        err = hipDeviceEnablePeerAccess(gpuFrom, 0);
        if (err == hipErrorPeerAccessAlreadyEnabled) {
            err = hipSuccess;
        }
        CUDA_CALL(err);
    }
}

float* allocateTestData(int gpu, size_t size, bool fill = false) {
    float* data = nullptr;
    CUDA_DEVICE(gpu);
    CUDA_CALL(hipMalloc((void**)&data, size));
    
    if(fill) {
        hiprandGenerator_t gen;
        CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
        CURAND_CALL(hiprandGenerateUniform(gen, data, size / sizeof(float)));
        CURAND_CALL(hiprandDestroyGenerator(gen));
    }
    
    return data;
}

void moveTestData(float* gpu1Data, float* gpu2Data, int gpuFrom, int gpuTo, size_t size) {
    enablePeerAccess(gpuFrom, gpuTo);
    
    CUDA_CALL(hipSetDevice(gpuFrom));
    
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    
    CUDA_CALL(hipEventRecord(start, 0));    
    CUDA_CALL(hipMemcpyPeer(gpu2Data, gpuTo, gpu1Data, gpuFrom, size));    
    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    float sizeInMB = size / (1024 * 1024);
    float sizeInGB = sizeInMB / 1024;
    float timeInSeconds = milliseconds / 1000.0;
    float speedGBps = sizeInGB / timeInSeconds;

    std::cout << "Copied " << (sizeInGB) << " GiB from GPU " << gpuFrom << " to GPU " 
        << gpuTo << " in " << formatDouble(milliseconds, 0) << "ms (" << formatDouble(speedGBps, 2)
        << " GB/s)" << std::endl;
    
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

}

void test(int gpu0, int gpu1) {

    const size_t dataSize = 1024 * 1024 * 1024;

    float* gpu0Data = allocateTestData(gpu0, dataSize, true);
    float* gpu1Data = allocateTestData(gpu1, dataSize, false);

    moveTestData(gpu0Data, gpu1Data, gpu0, gpu1, dataSize);

    CUDA_CALL(hipFree(gpu0Data));
    CUDA_CALL(hipFree(gpu1Data));
}

int main() {

    int num_devices = 0;
    hipGetDeviceCount(&num_devices);
    if (num_devices == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return -1;
    }
    for(int i = 0; i < num_devices; i++) {
        try{
            CUDA_CALL(hipSetDevice(i));
        }catch(std::runtime_error e) {
            std::cerr << "Error setting device " << i << ": " << e.what() << std::endl;
            return -1;
        }
    }

    std::cout << "Total devices found: " << num_devices << std::endl;
    for(int i = 0; i < num_devices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << std::endl << "[ Device " << i << ": " << prop.name << "]" << std::endl << std::endl;
        for(int j = 0; j < num_devices; j++) {
            if(i != j) {
                test(i, j);
            }
        }
    }
    return 0;
}